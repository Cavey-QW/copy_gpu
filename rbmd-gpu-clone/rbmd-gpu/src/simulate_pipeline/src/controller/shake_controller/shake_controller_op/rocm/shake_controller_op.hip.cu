#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "rbmd_define.h"
#include "shake_controller_op.h"
#include <math.h>
namespace op {
#define THREADS_PER_BLOCK 256

__device__ Real3 MinDistanceVec(const rbmd::Real& share_px_1,
                                const rbmd::Real& share_py_1, 
                                const rbmd::Real& share_pz_1, 
                                const rbmd::Real& share_px_2,
                                const rbmd::Real& share_py_2,
                                const rbmd::Real& share_pz_2,
                                Box* box)
{
    rbmd::Id periodicX = 1;
    rbmd::Id periodicY = 1;
    rbmd::Id periodicZ = 1;

    Real3 vec;
    vec.x = share_px_1 - share_px_2;
    vec.y = share_py_1 - share_py_2;
    vec.z = share_pz_1 - share_pz_2;
    
    // X
    if (periodicX)
    {
        if (ABS(vec.x) > box->_length[0] * 0.5)
        {
            vec.x -= (vec.x > 0 ? box->_length[0] : -(box->_length[0]));
        }
    }

    // Y
    if (periodicY)
    {
        if (ABS(vec.y) > box->_length[1] * 0.5)
        {
            vec.y -= (vec.y > 0 ? box->_length[1] : -(box->_length[1]));
        }
    }

    // Z
    if (periodicZ)
    {
        if (ABS(vec.z) > box->_length[2] * 0.5)
        {
            vec.z -= (vec.z > 0 ? box->_length[2] : -(box->_length[2]));
        }
    }

    return vec;
}

__device__ rbmd::Real Dot(const Real3& p_1,
                          const Real3& p_2)
    {
        return p_1.x * p_2.x + p_1.y * p_2.y + p_1.z * p_2.z;
    }

    __device__ rbmd::Real Abs(const rbmd::Real& value)
    {
        return value < 0 ? -value : value;
    }

    __device__ bool IsNan(const rbmd::Real& value) {
        //return value != value;  // TODO: isnan
        return (isnan(value) != 0);
}

__global__ void ShakeA(const rbmd::Id num_angle,
                       const rbmd::Real dt,
                       const rbmd::Real fmt2v,
                       Box* box,
                       const rbmd::Id* atom_id_to_idx,
                       const rbmd::Real* mass,
                       const rbmd::Id* atoms_type,
                       const Id3* angle_id_vec,
                       rbmd::Real* shake_px,
                       rbmd::Real* shake_py,
                       rbmd::Real* shake_pz,
                       rbmd::Real* shake_vx,
                       rbmd::Real* shake_vy,
                       rbmd::Real* shake_vz,
                       const rbmd::Real* fx,
                       const rbmd::Real* fy,
                       const rbmd::Real* fz,
                       rbmd::Id* flag_px,
                       rbmd::Id* flag_py,
                       rbmd::Id* flag_pz) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < num_angle) {

        rbmd::Id id_0x = angle_id_vec[tid].x;
        rbmd::Id id_1x = angle_id_vec[tid].y;
        rbmd::Id id_2x = angle_id_vec[tid].z;

    rbmd::Id id_0 = atom_id_to_idx[id_0x];
    rbmd::Id id_1 = atom_id_to_idx[id_1x];
    rbmd::Id id_2 = atom_id_to_idx[id_2x];

    //printf("tid为：%d----1输出结果为：%f, %f, %f \n %f, %f, %f \n  %f, %f, %f \n  %f, %f \n",tid,shake_px[id_0],shake_px[id_1],shake_px[id_2],shake_vx[id_0],shake_vx[id_1],shake_vx[id_2],
    //       mass[atoms_type[id_0]],mass[atoms_type[id_1]] ,mass[atoms_type[id_2]] , dt,fmt2v);

    Real3 shake_position_0, shake_position_1, shake_position_2;
    shake_position_0.x = shake_px[id_0] + shake_vx[id_0] * dt + 0.5 * dt * dt * fx[id_0] / mass[atoms_type[id_0]] * fmt2v;
    shake_position_0.y = shake_py[id_0] + shake_vy[id_0] * dt + 0.5 * dt * dt * fy[id_0] / mass[atoms_type[id_0]] * fmt2v;
    shake_position_0.z = shake_pz[id_0] + shake_vz[id_0] * dt + 0.5 * dt * dt * fz[id_0] / mass[atoms_type[id_0]] * fmt2v;

    shake_position_1.x = shake_px[id_1] + shake_vx[id_1] * dt + 0.5 * dt * dt * fx[id_1] / mass[atoms_type[id_1]] * fmt2v;
    shake_position_1.y = shake_py[id_1] + shake_vy[id_1] * dt + 0.5 * dt * dt * fy[id_1] / mass[atoms_type[id_1]] * fmt2v;
    shake_position_1.z = shake_pz[id_1] + shake_vz[id_1] * dt + 0.5 * dt * dt * fz[id_1] / mass[atoms_type[id_1]] * fmt2v;

    shake_position_2.x = shake_px[id_2] + shake_vx[id_2] * dt + 0.5 * dt * dt * fx[id_2] / mass[atoms_type[id_2]] * fmt2v;
    shake_position_2.y = shake_py[id_2] + shake_vy[id_2] * dt + 0.5 * dt * dt * fy[id_2] / mass[atoms_type[id_2]] * fmt2v;
    shake_position_2.z = shake_pz[id_2] + shake_vz[id_2] * dt + 0.5 * dt * dt * fz[id_2] / mass[atoms_type[id_2]] * fmt2v;

        //printf("tid为：%d----输出结果为：%f, %f, %f \n",tid,shake_position_0.x,shake_position_1.x,shake_position_2.x);


    rbmd::Real bond1 = 1.0;
    rbmd::Real bond2 = 1.0;
    rbmd::Real bond12 = SQRT(bond1 * bond1 + bond2 * bond2 - 2.0 * bond1 * bond2 * COS((109.4700 / 180.0) * M_PIf));
        //printf("tid为：%d----输出结果为：%f \n",tid,bond12);
    
    // minimum image
    Real3 r01 = MinDistanceVec(shake_px[id_1], shake_py[id_1], shake_pz[id_1], shake_px[id_0], shake_py[id_0], shake_pz[id_0], box);
    Real3 r12 = MinDistanceVec(shake_px[id_2], shake_py[id_2], shake_pz[id_2], shake_px[id_1], shake_py[id_1], shake_pz[id_1], box);
    Real3 r20 = MinDistanceVec(shake_px[id_0], shake_py[id_0], shake_pz[id_0], shake_px[id_2], shake_py[id_2], shake_pz[id_2], box);
        //printf("tid为：%d----输出结果为：%f, %f, %f \n",tid,r01.x,r12.x,r20.x);

    // s01,s02,s12 = distance vec after unconstrained update, with PBC
    Real3 s10 = MinDistanceVec(shake_position_0.x, shake_position_0.y, shake_position_0.z, shake_position_1.x, shake_position_1.y, shake_position_1.z, box);
    Real3 s21 = MinDistanceVec(shake_position_1.x, shake_position_1.y, shake_position_1.z, shake_position_2.x, shake_position_2.y, shake_position_2.z, box);
    Real3 s02 = MinDistanceVec(shake_position_2.x, shake_position_2.y, shake_position_2.z, shake_position_0.x, shake_position_0.y, shake_position_0.z, box);
        //printf("tid为：%d----输出结果为：%f, %f, %f \n",tid,s10.x,s21.x,s02.x);

    // scalar distances between atoms
    rbmd::Real r01sq = Dot(r01, r01);
    rbmd::Real r02sq = Dot(r20, r20);
    rbmd::Real r12sq = Dot(r12, r12);
    rbmd::Real s01sq = Dot(s10, s10);
    rbmd::Real s02sq = Dot(s02, s02);
    rbmd::Real s12sq = Dot(s21, s21);
       //printf("tid为：%d----s01sq输出结果为：%f, %f, %f \n",tid,r01sq,s01sq,s12sq);
    // matrix coeffs and rhs for lamda equations
    rbmd::Real invmass0 = 1 / mass[atoms_type[id_0]];
    rbmd::Real invmass1 = 1 / mass[atoms_type[id_1]];
    rbmd::Real invmass2 = 1 / mass[atoms_type[id_2]];
    rbmd::Real a11 = 2 * (invmass0 + invmass1) * Dot(s10, r01);
    rbmd::Real a12 = -2 * invmass1 * Dot(s10, r12);
    rbmd::Real a13 = -2 * invmass0 * Dot(s10, r20);
    rbmd::Real a21 = -2 * invmass1 * Dot(s21, r01);
    rbmd::Real a22 = 2 * (invmass1 + invmass2) * Dot(s21, r12);
    rbmd::Real a23 = -2 * invmass2 * Dot(s21, r20);
    rbmd::Real a31 = -2 * invmass0 * Dot(s02, r01);
    rbmd::Real a32 = -2 * invmass2 * Dot(s02, r12);
    rbmd::Real a33 = 2 * (invmass0 + invmass2) * (Dot(s02, r20));

      // inverse of matrix

      rbmd::Real determ = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 - a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;
      if (Abs(determ) < 0.0001)
      {
          printf("Shake determinant = 0.0");
      }


      rbmd::Real determinv = 1 / determ;

      rbmd::Real a11inv = determinv * (a22 * a33 - a23 * a32);
      rbmd::Real a12inv = -determinv * (a12 * a33 - a13 * a32);
      rbmd::Real a13inv = determinv * (a12 * a23 - a13 * a22);
      rbmd::Real a21inv = -determinv * (a21 * a33 - a23 * a31);
      rbmd::Real a22inv = determinv * (a11 * a33 - a13 * a31);
      rbmd::Real a23inv = -determinv * (a11 * a23 - a13 * a21);
      rbmd::Real a31inv = determinv * (a21 * a32 - a22 * a31);
      rbmd::Real a32inv = -determinv * (a11 * a32 - a12 * a31);
      rbmd::Real a33inv = determinv * (a11 * a22 - a12 * a21);


      rbmd::Real r0120 = Dot(r01, r20);
      rbmd::Real r0112 = Dot(r01, r12);
      rbmd::Real r2012 = Dot(r20, r12);

      rbmd::Real quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
      rbmd::Real quad1_1212 = invmass1 * invmass1 * r12sq;
      rbmd::Real quad1_2020 = invmass0 * invmass0 * r02sq;
      rbmd::Real quad1_0120 = -2 * (invmass0 + invmass1) * invmass0 * r0120;
      rbmd::Real quad1_0112 = -2 * (invmass0 + invmass1) * invmass1 * r0112;
      rbmd::Real quad1_2012 = 2 * invmass0 * invmass1 * r2012;

      rbmd::Real quad2_0101 = invmass1 * invmass1 * r01sq;
      rbmd::Real quad2_1212 = (invmass1 + invmass2) * (invmass1 + invmass2) * r12sq;
      rbmd::Real quad2_2020 = invmass2 * invmass2 * r02sq;
      rbmd::Real quad2_0120 = 2 * invmass1 * invmass2 * r0120;
      rbmd::Real quad2_0112 = -2 * (invmass1 + invmass2) * invmass1 * r0112;
      rbmd::Real quad2_2012 = -2 * (invmass1 + invmass2) * invmass2 * r2012;

      rbmd::Real quad3_0101 = invmass0 * invmass0 * r01sq;
      rbmd::Real quad3_1212 = invmass2 * invmass2 * r12sq;
      rbmd::Real quad3_2020 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
      rbmd::Real quad3_0120 = -2 * (invmass0 + invmass2) * invmass0 * r0120;
      rbmd::Real quad3_0112 = 2 * invmass0 * invmass2 * r0112;
      rbmd::Real quad3_2012 = -2 * (invmass0 + invmass2) * invmass2 * r2012;
        // printf("tid为：%d----倒数第6次quad1参数输出结果为：%f, %f, %f,%f, %f, %f,%f, %f, %f,%f, %f, %f,%f, %f, %f,%f, %f , %f, %f \n",tid,quad1_0101, quad1_1212 , quad1_2020,quad1_0120,quad1_0112,quad1_2012,quad2_0101
        //       ,quad2_1212,quad2_1212,quad2_2020,quad2_0120,quad2_0112,quad2_2012,quad3_0101,quad3_1212,quad3_2020,quad3_0120,quad3_0112,quad3_2012);
        // iterate until converged
        rbmd::Real tolerance = 0.00001;     // original 0.001
        rbmd::Id max_iter = 5000; // original: 100

        rbmd::Real lamda01 = 0.0;
        rbmd::Real lamda20 = 0.0;
        rbmd::Real lamda12 = 0.0;
        rbmd::Id niter = 0;
        rbmd::Id done = 0;
        rbmd::Id flag_overflow = 0;
        rbmd::Real quad1, quad2, quad3, b1, b2, b3, lamda01_new, lamda20_new, lamda12_new;

        while (!done && niter < max_iter)
        {

            quad1 = quad1_0101 * lamda01 * lamda01 + quad1_2020 * lamda20 * lamda20 +
                    quad1_1212 * lamda12 * lamda12 + quad1_0120 * lamda01 * lamda20 +
                    quad1_0112 * lamda01 * lamda12 + quad1_2012 * lamda20 * lamda12;

            quad2 = quad2_0101 * lamda01 * lamda01 + quad2_2020 * lamda20 * lamda20 +
                    quad2_1212 * lamda12 * lamda12 + quad2_0120 * lamda01 * lamda20 +
                    quad2_0112 * lamda01 * lamda12 + quad2_2012 * lamda20 * lamda12;

            quad3 = quad3_0101 * lamda01 * lamda01 + quad3_2020 * lamda20 * lamda20 +
                    quad3_1212 * lamda12 * lamda12 + quad3_0120 * lamda01 * lamda20 +
                    quad3_0112 * lamda01 * lamda12 + quad3_2012 * lamda20 * lamda12;

            b1 = bond1 * bond1 - s01sq - quad1;  // bond 是常值，s01sq应该 也可，
            b2 = bond2 * bond2 - s12sq - quad2;
            b3 = bond12 * bond12 - s02sq - quad3;
            //printf("tid为：%d----倒数第6次b参数输出结果为：%f, %f,%f, %f \n",tid,bond1, bond1 , s01sq , quad1);
            //printf("tid为：%d----倒数第6次b参数输出结果为：%f, %f,%f \n",tid,b1, b2 , b3);
            //printf("tid为：%d----倒数第6次quad参数输出结果为：%f, %f,%f \n",tid,quad1, quad2 , quad3);

            lamda01_new = a11inv * b1 + a12inv * b2 + a13inv * b3;
            lamda12_new = a21inv * b1 + a22inv * b2 + a23inv * b3;
            lamda20_new = a31inv * b1 + a32inv * b2 + a33inv * b3;
            //printf("tid为：%d----倒数第5次lamda01_new参数输出结果为：%f, %f,%f, %f,%f,%f \n",tid,a11inv , b1 , a12inv , b2 , a13inv , b3);

            done = 1;
            if (Abs(lamda01_new - lamda01) > tolerance)
                done = 0;
            if (Abs(lamda20_new - lamda20) > tolerance)
                done = 0;
            if (Abs(lamda12_new - lamda12) > tolerance)
                done = 0;

            lamda01 = lamda01_new;
            lamda20 = lamda20_new;
            lamda12 = lamda12_new;
           // printf("tid为：%d----倒数第4次lamda输出结果为：%f, %f \n",tid,lamda01, lamda20);

            if (IsNan(lamda01) || IsNan(lamda20) || IsNan(lamda12) ||
                Abs(lamda01) > 1e20 || Abs(lamda20) > 1e20 || Abs(lamda12) > 1e20)
            {
                done = 1;
                flag_overflow = 1;
            }
            niter++;
        }

        Real3 position_constraint_i0;
        Real3 position_constraint_i1;
        Real3 position_constraint_i2;

        position_constraint_i0.x = lamda01 * r01.x * invmass0 - lamda20 * r20.x * invmass0;
        position_constraint_i0.y = lamda01 * r01.y * invmass0 - lamda20 * r20.y * invmass0;
        position_constraint_i0.z = lamda01 * r01.z * invmass0 - lamda20 * r20.z * invmass0;

        position_constraint_i1.x = lamda12 * r12.x * invmass1 - lamda01 * r01.x * invmass1;
        position_constraint_i1.y = lamda12 * r12.y * invmass1 - lamda01 * r01.y * invmass1;
        position_constraint_i1.z = lamda12 * r12.z * invmass1 - lamda01 * r01.z * invmass1;

        position_constraint_i2.x = lamda20 * r20.x * invmass2 - lamda12 * r12.x * invmass2;
        position_constraint_i2.y = lamda20 * r20.y * invmass2 - lamda12 * r12.y * invmass2;
        position_constraint_i2.z = lamda20 * r20.z * invmass2 - lamda12 * r12.z * invmass2;
        //printf("tid为：%d----倒数第3次position_constraint_i0输出结果为：%f, %f, %f \n",tid,position_constraint_i0.x,position_constraint_i1.x,position_constraint_i2.x);
        //printf("tid为：%d----倒数第3次position_constraint_i0参数输出结果为：%f, %f, %f, %f, %f \n",tid,lamda01, r01.x , invmass0 , lamda20 , r20.x );



        Real3 velocity_constraint_i0,velocity_constraint_i1, velocity_constraint_i2;

        velocity_constraint_i0.x = position_constraint_i0.x / dt;
        velocity_constraint_i0.y = position_constraint_i0.y / dt;
        velocity_constraint_i0.z = position_constraint_i0.z / dt;

        velocity_constraint_i1.x = position_constraint_i1.x / dt;
        velocity_constraint_i1.y = position_constraint_i1.y / dt;
        velocity_constraint_i1.z = position_constraint_i1.z / dt;

        velocity_constraint_i2.x = position_constraint_i2.x / dt;
        velocity_constraint_i2.y = position_constraint_i2.y / dt;
        velocity_constraint_i2.z = position_constraint_i2.z / dt;
        //printf("tid为：%d----倒数第二次velocity_constraint输出结果为：%f, %f, %f \n",tid,velocity_constraint_i0.x,velocity_constraint_i1.x,velocity_constraint_i2.x);
        Real3 shake_velocity_0,shake_velocity_1,shake_velocity_2;
        shake_velocity_0.x = shake_vx[id_0] + 0.5 * dt * fx[id_0]/mass[atoms_type[id_0]] * fmt2v;
        shake_velocity_0.y = shake_vy[id_0] + 0.5 * dt * fy[id_0]/mass[atoms_type[id_0]] * fmt2v;
        shake_velocity_0.z = shake_vz[id_0] + 0.5 * dt * fz[id_0]/mass[atoms_type[id_0]] * fmt2v;

        shake_velocity_1.x = shake_vx[id_1] + 0.5 * dt * fx[id_1]/mass[atoms_type[id_1]] * fmt2v;
        shake_velocity_1.y = shake_vy[id_1] + 0.5 * dt * fy[id_1]/mass[atoms_type[id_1]] * fmt2v;
        shake_velocity_1.z = shake_vz[id_1] + 0.5 * dt * fz[id_1]/mass[atoms_type[id_1]] * fmt2v;

        shake_velocity_2.x = shake_vx[id_2] + 0.5 * dt * fx[id_2]/mass[atoms_type[id_2]] * fmt2v;
        shake_velocity_2.y = shake_vy[id_2] + 0.5 * dt * fy[id_2]/mass[atoms_type[id_2]] * fmt2v;
        shake_velocity_2.z = shake_vz[id_2] + 0.5 * dt * fz[id_2]/mass[atoms_type[id_2]] * fmt2v;
        //printf("tid为：%d----倒数第三次输出结果为：%f, %f, %f \n",tid,shake_px[id_0],shake_px[id_1],shake_px[id_2]);

        // velocity
        shake_vx[id_0] = shake_velocity_0.x + velocity_constraint_i0.x;
        shake_vy[id_0] = shake_velocity_0.y + velocity_constraint_i0.y;
        shake_vz[id_0] = shake_velocity_0.z + velocity_constraint_i0.z;

        shake_vx[id_1] = shake_velocity_1.x + velocity_constraint_i1.x;
        shake_vy[id_1] = shake_velocity_1.y + velocity_constraint_i1.y;
        shake_vz[id_1] = shake_velocity_1.z + velocity_constraint_i1.z;

        shake_vx[id_2] = shake_velocity_2.x + velocity_constraint_i2.x;
        shake_vy[id_2] = shake_velocity_2.y + velocity_constraint_i2.y;
        shake_vz[id_2] = shake_velocity_2.z + velocity_constraint_i2.z;

        // position
        shake_px[id_0] = shake_position_0.x + position_constraint_i0.x;
        shake_py[id_0] = shake_position_0.y + position_constraint_i0.y;
        shake_pz[id_0] = shake_position_0.z + position_constraint_i0.z;

        shake_px[id_1] = shake_position_1.x + position_constraint_i1.x;
        shake_py[id_1] = shake_position_1.y + position_constraint_i1.y;
        shake_pz[id_1] = shake_position_1.z + position_constraint_i1.z;

        shake_px[id_2] = shake_position_2.x + position_constraint_i2.x;
        shake_py[id_2] = shake_position_2.y + position_constraint_i2.y;
        shake_pz[id_2] = shake_position_2.z + position_constraint_i2.z;
        //printf("tid为：%d----倒数第二次输出结果为：%f, %f, %f \n",tid,shake_px[id_0],shake_px[id_1],shake_px[id_2]);
        //printf("tid为：%d----倒数第二次shake_position输出结果为：%f, %f, %f \n",tid,shake_position_0.x,shake_position_1.x,shake_position_2.x);
        //printf("tid为：%d----倒数第二次shake_position输出结果为：%f, %f, %f \n",tid,position_constraint_i0.x,position_constraint_i1.x,position_constraint_i2.x);

        // pbc
        Real3 whole_position_pbc_0, whole_position_pbc_1, whole_position_pbc_2;

        // position
        whole_position_pbc_0.x = shake_px[id_0];
        whole_position_pbc_0.y = shake_py[id_0];
        whole_position_pbc_0.z = shake_pz[id_0];

        whole_position_pbc_1.x = shake_px[id_1];
        whole_position_pbc_1.y = shake_py[id_1];
        whole_position_pbc_1.z = shake_pz[id_1];

        whole_position_pbc_2.x = shake_px[id_2];
        whole_position_pbc_2.y = shake_py[id_2];
        whole_position_pbc_2.z = shake_pz[id_2];

        //position_flag
        Id3 whole_pts_flag_pbc_0, whole_pts_flag_pbc_1, whole_pts_flag_pbc_2;

        whole_pts_flag_pbc_0.x = flag_px[id_0];
        whole_pts_flag_pbc_0.y = flag_py[id_0];
        whole_pts_flag_pbc_0.z = flag_pz[id_0];

        whole_pts_flag_pbc_1.x = flag_px[id_1];
        whole_pts_flag_pbc_1.y = flag_py[id_1];
        whole_pts_flag_pbc_1.z = flag_pz[id_1];

        whole_pts_flag_pbc_2.x = flag_px[id_2];
        whole_pts_flag_pbc_2.y = flag_py[id_2];
        whole_pts_flag_pbc_2.z = flag_pz[id_2];

        // whole_position_pbc_0  min
        if (whole_position_pbc_0.x < box->_coord_min[0])
        {
            whole_position_pbc_0.x += (box->_coord_max[0] - box->_coord_min[0]);
            whole_pts_flag_pbc_0.x -= 1;
        }
        if (whole_position_pbc_0.y < box->_coord_min[1])
        {
            whole_position_pbc_0.y += (box->_coord_max[1] - box->_coord_min[1]);
            whole_pts_flag_pbc_0.y -= 1;
        }
        if (whole_position_pbc_0.z < box->_coord_min[2])
        {
            whole_position_pbc_0.z += (box->_coord_max[2] - box->_coord_min[2]);
            whole_pts_flag_pbc_0.z -= 1;
        }

        // whole_position_pbc_0  max
        if (whole_position_pbc_0.x > box->_coord_max[0])
        {
            whole_position_pbc_0.x -= (box->_coord_max[0] - box->_coord_min[0]);
            whole_pts_flag_pbc_0.x += 1;
        }
        if (whole_position_pbc_0.y > box->_coord_max[1])
        {
            whole_position_pbc_0.y -= (box->_coord_max[1] - box->_coord_min[1]);
            whole_pts_flag_pbc_0.y += 1;
        }
        if (whole_position_pbc_0.z > box->_coord_max[2])
        {
            whole_position_pbc_0.z -= (box->_coord_max[2] - box->_coord_min[2]);
            whole_pts_flag_pbc_0.z += 1;
        }

        // whole_position_pbc_1  min
        if (whole_position_pbc_1.x < box->_coord_min[0])
        {
            whole_position_pbc_1.x += (box->_coord_max[0] - box->_coord_min[0]);
            whole_pts_flag_pbc_1.x -= 1;
        }
        if (whole_position_pbc_1.y < box->_coord_min[1])
        {
            whole_position_pbc_1.y += (box->_coord_max[1] - box->_coord_min[1]);
            whole_pts_flag_pbc_1.y -= 1;
        }
        if (whole_position_pbc_1.z < box->_coord_min[2])
        {
            whole_position_pbc_1.z += (box->_coord_max[2] - box->_coord_min[2]);
            whole_pts_flag_pbc_1.z -= 1;
        }

        // whole_position_pbc_1  max
        if (whole_position_pbc_1.x > box->_coord_max[0])
        {
            whole_position_pbc_1.x -= (box->_coord_max[0] - box->_coord_min[0]);
            whole_pts_flag_pbc_1.x += 1;
        }
        if (whole_position_pbc_1.y > box->_coord_max[1])
        {
            whole_position_pbc_1.y -= (box->_coord_max[1] - box->_coord_min[1]);
            whole_pts_flag_pbc_1.y += 1;
        }
        if (whole_position_pbc_1.z > box->_coord_max[2])
        {
            whole_position_pbc_1.z -= (box->_coord_max[2] - box->_coord_min[2]);
            whole_pts_flag_pbc_1.z += 1;
        }

        // whole_position_pbc_2  min
        if (whole_position_pbc_2.x < box->_coord_min[0])
        {
            whole_position_pbc_2.x += (box->_coord_max[0] - box->_coord_min[0]);
            whole_pts_flag_pbc_2.x -= 1;
        }
        if (whole_position_pbc_2.y < box->_coord_min[1])
        {
            whole_position_pbc_2.y += (box->_coord_max[1] - box->_coord_min[1]);
            whole_pts_flag_pbc_2.y -= 1;
        }
        if (whole_position_pbc_2.z < box->_coord_min[2])
        {
            whole_position_pbc_2.z += (box->_coord_max[2] - box->_coord_min[2]);
            whole_pts_flag_pbc_2.z -= 1;
        }

        // whole_position_pbc_2  max
        if (whole_position_pbc_2.x > box->_coord_max[0])
        {
            whole_position_pbc_2.x -= (box->_coord_max[0] - box->_coord_min[0]);
            whole_pts_flag_pbc_2.x += 1;
        }
        if (whole_position_pbc_2.y > box->_coord_max[1])
        {
            whole_position_pbc_2.y -= (box->_coord_max[1] - box->_coord_min[1]);
            whole_pts_flag_pbc_2.y += 1;
        }
        if (whole_position_pbc_2.z > box->_coord_max[2])
        {
            whole_position_pbc_2.z -= (box->_coord_max[2] - box->_coord_min[2]);
            whole_pts_flag_pbc_2.z += 1;
        }

        // pts
        shake_px[id_0] = whole_position_pbc_0.x;
        shake_px[id_1] = whole_position_pbc_0.y;
        shake_px[id_2] = whole_position_pbc_0.z;
        flag_px[id_0] = whole_pts_flag_pbc_0.x;
        flag_px[id_1] = whole_pts_flag_pbc_0.y;
        flag_px[id_2] = whole_pts_flag_pbc_0.z;

        shake_py[id_0] = whole_position_pbc_1.x;
        shake_py[id_1] = whole_position_pbc_1.y;
        shake_py[id_2] = whole_position_pbc_1.z;
        flag_py[id_0] = whole_pts_flag_pbc_1.x;
        flag_py[id_1] = whole_pts_flag_pbc_1.y;
        flag_py[id_2] = whole_pts_flag_pbc_1.z;

        shake_pz[id_0] = whole_position_pbc_2.x;
        shake_pz[id_1] = whole_position_pbc_2.y;
        shake_pz[id_2] = whole_position_pbc_2.z;
        flag_pz[id_0] = whole_pts_flag_pbc_2.x;
        flag_pz[id_1] = whole_pts_flag_pbc_2.y;
        flag_pz[id_2] = whole_pts_flag_pbc_2.z;

  }
}

__global__ void ShakeB(const rbmd::Id num_angle,
                       const rbmd::Real dt,
                       const rbmd::Real fmt2v,
                       Box* box,
                       const rbmd::Id* atom_id_to_idx,
                       const rbmd::Real* mass,
                       const rbmd::Id* atoms_type,
                       const Id3* angle_id_vec,
                       rbmd::Real* px,
                       rbmd::Real* py,
                       rbmd::Real* pz,
                       rbmd::Real* shake_vx,
                       rbmd::Real* shake_vy,
                       rbmd::Real* shake_vz,
                       const rbmd::Real* fx,
                       const rbmd::Real* fy,
                       const rbmd::Real* fz) {

        int tid = threadIdx.x + blockIdx.x * blockDim.x;

        if (tid < num_angle) {
            rbmd::Id id_0x = angle_id_vec[tid].x;
            rbmd::Id id_1x = angle_id_vec[tid].y;
            rbmd::Id id_2x = angle_id_vec[tid].z;

            rbmd::Id id_0 = atom_id_to_idx[id_0x];
            rbmd::Id id_1 = atom_id_to_idx[id_1x];
            rbmd::Id id_2 = atom_id_to_idx[id_2x];

            Real3 shake_velocity_0,shake_velocity_1,shake_velocity_2;
            shake_velocity_0.x = shake_vx[id_0] + 0.5 * dt * fx[id_0]/mass[atoms_type[id_0]] * fmt2v;
            shake_velocity_0.y = shake_vy[id_0] + 0.5 * dt * fy[id_0]/mass[atoms_type[id_0]] * fmt2v;
            shake_velocity_0.z = shake_vz[id_0] + 0.5 * dt * fz[id_0]/mass[atoms_type[id_0]] * fmt2v;

            shake_velocity_1.x = shake_vx[id_1] + 0.5 * dt * fx[id_1]/mass[atoms_type[id_1]] * fmt2v;
            shake_velocity_1.y = shake_vy[id_1] + 0.5 * dt * fy[id_1]/mass[atoms_type[id_1]] * fmt2v;
            shake_velocity_1.z = shake_vz[id_1] + 0.5 * dt * fz[id_1]/mass[atoms_type[id_1]] * fmt2v;

            shake_velocity_2.x = shake_vx[id_2] + 0.5 * dt * fx[id_2]/mass[atoms_type[id_2]] * fmt2v;
            shake_velocity_2.y = shake_vy[id_2] + 0.5 * dt * fy[id_2]/mass[atoms_type[id_2]] * fmt2v;
            shake_velocity_2.z = shake_vz[id_2] + 0.5 * dt * fz[id_2]/mass[atoms_type[id_2]] * fmt2v;

            // minimum image
            Real3 r01 = MinDistanceVec(px[id_1], py[id_1], pz[id_1], px[id_0], py[id_0], pz[id_0], box);
            Real3 r12 = MinDistanceVec(px[id_2], py[id_2], pz[id_2], px[id_1], py[id_1], pz[id_1], box);
            Real3 r20 = MinDistanceVec(px[id_0], py[id_0], pz[id_0], px[id_2], py[id_2], pz[id_2], box);

            //rbmd::Real sv10[3],sv21[3],sv02[3];
            Real3 sv10, sv21,sv02;
            sv10.x = shake_velocity_0.x - shake_velocity_1.x;
            sv10.y = shake_velocity_0.y - shake_velocity_1.y;
            sv10.z = shake_velocity_0.z - shake_velocity_1.z;

            sv21.x = shake_velocity_1.x - shake_velocity_2.x;
            sv21.y = shake_velocity_1.y - shake_velocity_2.y;
            sv21.z = shake_velocity_1.z - shake_velocity_2.z;

            sv02.x = shake_velocity_2.x - shake_velocity_0.x;
            sv02.y = shake_velocity_2.y - shake_velocity_0.y;
            sv02.z = shake_velocity_2.z - shake_velocity_0.z;

            rbmd::Real invmass0 = 1 / mass[atoms_type[id_0]];
            rbmd::Real invmass1 = 1 / mass[atoms_type[id_1]];
            rbmd::Real invmass2 = 1 / mass[atoms_type[id_2]];

            Real3 c, l;
            Real3 a_0, a_1, a_2;

            // setup matrix
            a_0.x = (invmass1 + invmass0) * Dot(r01, r01);
            a_0.y = -invmass1 * Dot(r01, r12);
            a_0.z = (-invmass0) * Dot(r01, r20);
            a_1.x = a_0.y;
            a_1.y = (invmass1 + invmass2) * Dot(r12, r12);
            a_1.z = -(invmass2) * Dot(r20, r12);
            a_2.x = a_0.z;
            a_2.y = a_1.z;
            a_2.z = (invmass0 + invmass2) * Dot(r20, r20);

            // sestup RHS
            c.x = -Dot(sv10, r01);
            c.y = -Dot(sv21, r12);
            c.z = -Dot(sv02, r20);

            Real3 ai_0, ai_1, ai_2;
            rbmd::Real determ, determinv = 0.0;

            // calculate the determinant of the matrix
            determ = a_0.x * a_1.y * a_2.z + a_0.y * a_1.z * a_2.x + a_0.z * a_1.x * a_2.y - a_0.x * a_1.z * a_2.y - a_0.y * a_1.x * a_2.z -
                     a_0.z * a_1.y * a_2.x;

            // check if matrix is actually invertible
            if (Abs(determ) < 0.0001)
                printf(" Error: Rattle determinant = 0.0 ");

            // calculate the inverse 3x3 matrix: A^(-1) = (ai_jk)
            determinv = 1 / determ;
            ai_0.x =  determinv * (a_1.y * a_2.z - a_1.z * a_2.y);
            ai_0.y = -determinv * (a_0.y * a_2.z - a_0.z * a_2.y);
            ai_0.z =  determinv * (a_0.y * a_1.z - a_0.z * a_1.y);
            ai_1.x = -determinv * (a_1.x * a_2.z - a_1.z * a_2.x);
            ai_1.y =  determinv * (a_0.x * a_2.z - a_0.z * a_2.x);
            ai_1.z = -determinv * (a_0.x * a_1.z - a_0.z * a_1.x);
            ai_2.x =  determinv * (a_1.x * a_2.y - a_1.y * a_2.x);
            ai_2.y = -determinv * (a_0.x * a_2.y - a_0.y * a_2.x);
            ai_2.z =  determinv * (a_0.x * a_1.y - a_0.y * a_1.x);

            // calculate the solution:  (l01, l02, l12)^T = A^(-1) * c
            l.x = 0;
            l.y = 0;
            l.z = 0;

            l.x += ai_0.x * c.x;
            l.x += ai_0.y * c.y;
            l.x += ai_0.z * c.z;

            l.y += ai_1.x * c.x;
            l.y += ai_1.y * c.y;
            l.y += ai_1.z * c.z;

            l.z += ai_2.x * c.x;
            l.z += ai_2.y * c.y;
            l.z += ai_2.z * c.z;

            // [l01,l02,l12]^T = [lamda12,lamda23,lamda31]^T
            Real3 velocity_constraint_i0, velocity_constraint_i1, velocity_constraint_i2;
            velocity_constraint_i0.x = l.x * r01.x * invmass0 - l.z * r20.x * invmass0;
            velocity_constraint_i0.y = l.x * r01.y * invmass0 - l.z * r20.y * invmass0;
            velocity_constraint_i0.z = l.x * r01.z * invmass0 - l.z * r20.z * invmass0;

            velocity_constraint_i1.x = l.y * r12.x * invmass0 - l.x * r01.x * invmass0;
            velocity_constraint_i1.y = l.y * r12.y * invmass0 - l.x * r01.y * invmass0;
            velocity_constraint_i1.z = l.y * r12.z * invmass0 - l.x * r01.z * invmass0;

            velocity_constraint_i2.x = l.z * r20.x * invmass0 - l.y * r12.x * invmass0;
            velocity_constraint_i2.y = l.z * r20.y * invmass0 - l.y * r12.y * invmass0;
            velocity_constraint_i2.z = l.z * r20.z * invmass0 - l.y * r12.z * invmass0;

            shake_vx[id_0] = shake_velocity_0.x + velocity_constraint_i0.x;
            shake_vy[id_0] = shake_velocity_0.y + velocity_constraint_i0.y;
            shake_vz[id_0] = shake_velocity_0.z + velocity_constraint_i0.z;

            shake_vx[id_1] = shake_velocity_1.x + velocity_constraint_i1.x;
            shake_vy[id_1] = shake_velocity_1.y + velocity_constraint_i1.y;
            shake_vz[id_1] = shake_velocity_1.z + velocity_constraint_i1.z;

            shake_vx[id_2] = shake_velocity_2.x + velocity_constraint_i2.x;
            shake_vy[id_2] = shake_velocity_2.y + velocity_constraint_i2.y;
            shake_vz[id_2] = shake_velocity_2.z + velocity_constraint_i2.z;

            Real3 a00, a01, a02, a10, a11, a12, a20, a21, a22, cal_v12t, cal_v23t, cal_v31t;
            a00.x=(invmass1 + invmass0) * r01.x;
            a00.y=(invmass1 + invmass0) * r01.y;
            a00.z=(invmass1 + invmass0) * r01.z;

            a01.x = -invmass1 * r12.x;
            a01.y = -invmass1 * r12.y;
            a01.z = -invmass1 * r12.z;

            a02.x = (-invmass0) * r20.x;
            a02.y = (-invmass0) * r20.y;
            a02.z = (-invmass0) * r20.z;

            a10.x = -invmass1 * r01.x;
            a10.y = -invmass1 * r01.y;
            a10.z = -invmass1 * r01.z;

            a11.x = (invmass1 + invmass2) * r12.x;
            a11.y = (invmass1 + invmass2) * r12.y;
            a11.z = (invmass1 + invmass2) * r12.z;

            a12.x = -(invmass2) * r20.x;
            a12.y = -(invmass2) * r20.y;
            a12.z = -(invmass2) * r20.z;

            a20.x = (-invmass0) * r01.x;
            a20.y = (-invmass0) * r01.y;
            a20.z = (-invmass0) * r01.z;

            a21.x = -(invmass2) * r12.x;
            a21.y = -(invmass2) * r12.y;
            a21.z = -(invmass2) * r12.z;

            a22.x = (invmass0 + invmass2) * r20.x;
            a22.y = (invmass0 + invmass2) * r20.y;
            a22.z = (invmass0 + invmass2) * r20.z;

            cal_v12t.x = sv10.x + (a00.x * l.x + a01.x * l.y + a02.x * l.z);
            cal_v12t.y = sv10.y + (a00.y * l.x + a01.y * l.y + a02.y * l.z);
            cal_v12t.z = sv10.z + (a00.z * l.x + a01.z * l.y + a02.z * l.z);

            cal_v23t.x = sv21.x + (a10.x * l.x + a11.x * l.y + a12.x * l.z);
            cal_v23t.y = sv21.y + (a10.y * l.x + a11.y * l.y + a12.y * l.z);
            cal_v23t.z = sv21.z + (a10.z * l.x + a11.z * l.y + a12.z * l.z);

            cal_v31t.x = sv02.x + (a20.x * l.x + a21.x * l.y + a22.x * l.z);
            cal_v31t.y = sv02.y + (a20.y * l.x + a21.y * l.y + a22.y * l.z);
            cal_v31t.z = sv02.z + (a20.z * l.x + a21.z * l.y + a22.z * l.z);


            rbmd::Real cal_dv1 = Dot(r01, cal_v12t);
            rbmd::Real cal_dv2 = Dot(r20, cal_v31t);
            rbmd::Real cal_dv12 = Dot(r12, cal_v23t);

            Real3 velocity01, velocity20, velocity12;
            velocity01.x = shake_vx[id_1] - shake_vx[id_0];
            velocity01.y = shake_vy[id_1] - shake_vy[id_0];
            velocity01.z = shake_vz[id_1] - shake_vz[id_0];

            velocity20.x = shake_vx[id_0] - shake_vx[id_2];
            velocity20.y = shake_vy[id_0] - shake_vy[id_2];
            velocity20.z = shake_vz[id_0] - shake_vz[id_2];

            velocity12.x = shake_vx[id_2] - shake_vx[id_1];
            velocity12.y = shake_vy[id_2] - shake_vy[id_1];
            velocity12.z = shake_vz[id_2] - shake_vz[id_1];

            rbmd::Real dv1 = Dot(r01, velocity01);
            rbmd::Real dv2 = Dot(r20, velocity20);
            rbmd::Real dv12 = Dot(r12, velocity12);

            if (Abs(dv1) > 0.1 || Abs(dv2) > 0.1 || Abs(dv12) > 0.1)
            {
                printf("i0 = %d, i1 = %d, i2 = %d\n", id_0, id_1, id_2);
                printf("dv1 = %f, dv2 = %f, dv12 = %f\n", dv1, dv2, dv12);
                printf("cal_dv1 = %f, cal_dv2 = %f, cal_dv12 = %f\n", cal_dv1, cal_dv2, cal_dv12);
                printf("velocity_i0 = [%f,%f,%f], velocity_i1 = [%f,%f,%f], velocity_i2 = [%f,%f,%f]\n",
                       shake_vx[id_0], shake_vy[id_0], shake_vz[id_0],
                       shake_vx[id_1], shake_vy[id_1], shake_vz[id_1],
                       shake_vx[id_2], shake_vy[id_2], shake_vz[id_2]);
                printf("\n");
            }
        }
    }

void ShakeAOp<device::DEVICE_GPU>::operator()(const rbmd::Id num_angle,
                                              const rbmd::Real dt,
                                              const rbmd::Real fmt2v,
                                              Box* box,
                                              const rbmd::Id* atom_id_to_idx,
                                              const rbmd::Real* mass,
                                              const rbmd::Id* atoms_type,
                                              const Id3* angle_id_vec,
                                              rbmd::Real* shake_px,
                                              rbmd::Real* shake_py,
                                              rbmd::Real* shake_pz,
                                              rbmd::Real* shake_vx,
                                              rbmd::Real* shake_vy,
                                              rbmd::Real* shake_vz,
                                              const rbmd::Real* fx,
                                              const rbmd::Real* fy,
                                              const rbmd::Real* fz,
                                              rbmd::Id* flag_px,
                                              rbmd::Id* flag_py,
                                              rbmd::Id* flag_pz) 
{
  unsigned int blocks_per_grid = (num_angle + BLOCK_SIZE - 1) / BLOCK_SIZE;
  CHECK_KERNEL(ShakeA<<<blocks_per_grid, BLOCK_SIZE, 0, 0>>>(
      num_angle, dt, fmt2v, box, atom_id_to_idx,mass, atoms_type, angle_id_vec,shake_px, shake_py, shake_pz, shake_vx, shake_vy, shake_vz, fx, fy, fz, flag_px, flag_py, flag_pz));
}

    void ShakeBOp<device::DEVICE_GPU>::operator()(const rbmd::Id num_angle,
                                                  const rbmd::Real dt,
                                                  const rbmd::Real fmt2v,
                                                  Box* box,
                                                  const rbmd::Id* atom_id_to_idx,
                                                  const rbmd::Real* mass,
                                                  const rbmd::Id* atoms_type,
                                                  const Id3* angle_id_vec,
                                                  rbmd::Real* px,
                                                  rbmd::Real* py,
                                                  rbmd::Real* pz,
                                                  rbmd::Real* shake_vx,
                                                  rbmd::Real* shake_vy,
                                                  rbmd::Real* shake_vz,
                                                  const rbmd::Real* fx,
                                                  const rbmd::Real* fy,
                                                  const rbmd::Real* fz)
    {
        unsigned int blocks_per_grid = (num_angle + BLOCK_SIZE - 1) / BLOCK_SIZE;
        CHECK_KERNEL(ShakeB<<<blocks_per_grid, BLOCK_SIZE, 0, 0>>>(num_angle, dt, fmt2v, box, atom_id_to_idx, mass, atoms_type,
                                                                   angle_id_vec,px, py, pz, shake_vx, shake_vy, shake_vz, fx, fy, fz));
    }
}  // namespace op
